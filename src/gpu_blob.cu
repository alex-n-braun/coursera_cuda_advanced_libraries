#include <hip/hip_runtime.h>

#include <iostream>

#include "gpu_blob.hpp"

GpuBlob::GpuBlob(std::size_t size) : m_size(size), m_data(nullptr) {
    hipError_t err = hipMalloc(&m_data, size);
    if (err != hipSuccess) {
        throw std::runtime_error("Failed to allocate device memory at " + std::string(__FILE__) +
                                 ":" + std::to_string(__LINE__));
    }
}
GpuBlob::~GpuBlob() {
    hipError_t err = hipFree(m_data);
    if (err != hipSuccess) {
        std::cerr << "Failed to free device memory at " << __FILE__ << ":" << __LINE__ << '\n';
        std::terminate();
    }
}
void GpuBlob::copyFrom(const void* data) {
    hipError_t err = hipMemcpy(m_data, data, m_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        throw std::runtime_error("Failed to copy data to device at " + std::string(__FILE__) + ":" +
                                 std::to_string(__LINE__));
    }
}
void GpuBlob::copyTo(void* data) const {
    hipError_t err = hipMemcpy(data, m_data, m_size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        throw std::runtime_error("Failed to copy data from device at " + std::string(__FILE__) +
                                 ":" + std::to_string(__LINE__));
    }
}
void* GpuBlob::data() { return m_data; }
const void* GpuBlob::data() const { return m_data; }
std::size_t GpuBlob::size() const { return m_size; }
