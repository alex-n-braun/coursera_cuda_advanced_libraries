#include "cudaKernels.hpp"
#include "imageManip.hpp"

void convertUint8ToFloat(ImageGPU<float, 4>& output, const ImageGPU<std::uint8_t, 4>& input,
                         const hipStream_t& stream) {
    kernelConvertUint8ToFloat<<<(input.size() + 255) / 256, 256, 0, stream>>>(
        input.data(), output.data(), input.size());

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw std::runtime_error("CUDA error (kernelConvertUint8ToFloat): " +
                                 std::string(hipGetErrorString(err)));
    }
}

void convertFloatToUint8(ImageGPU<std::uint8_t, 4>& output, const ImageGPU<float, 4>& input,
                         const hipStream_t& stream) {
    kernelConvertFloatToUint8<<<(input.size() + 255) / 256, 256, 0, stream>>>(
        input.data(), output.data(), input.size());

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw std::runtime_error("CUDA error (kernelConvertFloatToUint8): " +
                                 std::string(hipGetErrorString(err)));
    }
}

void setChannel(ImageGPU<float, 4>& data, int channel, float value, const hipStream_t& stream) {
    kernelSetChannel<<<(data.size() + 255) / 256, 256, 0, stream>>>(data.data(), channel, value, 4,
                                                                    data.numPixels());

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw std::runtime_error("CUDA error (kernelSetChannel): " +
                                 std::string(hipGetErrorString(err)));
    }
}

template <std::size_t Channels>
void pointwiseAbs_(ImageGPU<float, Channels>& output, const ImageGPU<float, Channels>& input,
                   const hipStream_t& stream) {
    kernelPointwiseAbs<<<(input.size() + 255) / 256, 256, 0, stream>>>(input.data(), output.data(),
                                                                       input.size());

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw std::runtime_error("CUDA error (kernelPointwiseAbs): " +
                                 std::string(hipGetErrorString(err)));
    }
}

template <>
void pointwiseAbs<2>(ImageGPU<float, 2>& output, const ImageGPU<float, 2>& input,
                     const hipStream_t& stream) {
    pointwiseAbs_<2>(output, input, stream);
}

template <std::size_t Channels>
void pointwiseMin_(ImageGPU<float, Channels>& output, float minValue,
                   const ImageGPU<float, Channels>& input, const hipStream_t& stream) {
    kernelPointwiseMin<<<(input.size() + 255) / 256, 256, 0, stream>>>(input.data(), minValue,
                                                                       output.data(), input.size());

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw std::runtime_error("CUDA error (kernelPointwiseMin): " +
                                 std::string(hipGetErrorString(err)));
    }
}

template <>
void pointwiseMin<1>(ImageGPU<float, 1>& output, float minValue, const ImageGPU<float, 1>& input,
                     const hipStream_t& stream) {
    pointwiseMin_<1>(output, minValue, input, stream);
}

template <std::size_t Channels>
void pointwiseHalo_(ImageGPU<float, Channels>& output, const ImageGPU<float, Channels>& rgbInput,
                    const ImageGPU<float, Channels>& haloInput, const hipStream_t& stream) {
    if (rgbInput.size() != haloInput.size() || rgbInput.size() != output.size()) {
        throw std::runtime_error("Image sizes do not match");
    }

    kernelPointwiseHalo<<<(rgbInput.size() + 255) / 256, 256, 0, stream>>>(
        rgbInput.data(), haloInput.data(), output.data(), rgbInput.size());

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw std::runtime_error("CUDA error (kernelPointwiseHalo): " +
                                 std::string(hipGetErrorString(err)));
    }
}

template <>
void pointwiseHalo<4>(ImageGPU<float, 4>& output, const ImageGPU<float, 4>& rgbInput,
                      const ImageGPU<float, 4>& haloInput, const hipStream_t& stream) {
    pointwiseHalo_<4>(output, rgbInput, haloInput, stream);
}
